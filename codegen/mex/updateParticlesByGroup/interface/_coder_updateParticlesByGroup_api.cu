//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_updateParticlesByGroup_api.cu
//
// Code generation for function '_coder_updateParticlesByGroup_api'
//

// Include files
#include "_coder_updateParticlesByGroup_api.h"
#include "rt_nonfinite.h"
#include "updateParticlesByGroup.h"
#include "updateParticlesByGroup_data.h"
#include "updateParticlesByGroup_types.h"

// Function Declarations
static int32_T (*b_emlrt_marshallIn(const mxArray *b_nullptr,
                                    const char_T *identifier))[1000];

static int32_T (*b_emlrt_marshallIn(const mxArray *u,
                                    const emlrtMsgIdentifier *parentId))[1000];

static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[36]);

static real_T c_emlrt_marshallIn(const mxArray *b_nullptr,
                                 const char_T *identifier);

static real_T c_emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId);

static real_T (*d_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[3000];

static int32_T (*e_emlrt_marshallIn(const mxArray *src,
                                    const emlrtMsgIdentifier *msgId))[1000];

static real_T (*emlrt_marshallIn(const mxArray *b_nullptr,
                                 const char_T *identifier))[3000];

static real_T (*emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId))[3000];

static void emlrt_marshallIn(const mxArray *b_nullptr, const char_T *identifier,
                             cell_wrap_0 y[5]);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             cell_wrap_0 y[5]);

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, real_T y[36]);

static void emlrt_marshallOut(const real_T u[3000], const mxArray *y);

static real_T f_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId);

// Function Definitions
static int32_T (*b_emlrt_marshallIn(const mxArray *b_nullptr,
                                    const char_T *identifier))[1000]
{
  emlrtMsgIdentifier thisId;
  int32_T(*y)[1000];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = b_emlrt_marshallIn(emlrtAlias(b_nullptr), &thisId);
  emlrtDestroyArray(&b_nullptr);
  return y;
}

static int32_T (*b_emlrt_marshallIn(const mxArray *u,
                                    const emlrtMsgIdentifier *parentId))[1000]
{
  int32_T(*y)[1000];
  y = e_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static void b_emlrt_marshallIn(const mxArray *src,
                               const emlrtMsgIdentifier *msgId, real_T ret[36])
{
  static const int32_T dims[2]{6, 6};
  real_T(*r)[36];
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 2U,
                          (const void *)&dims[0]);
  r = (real_T(*)[36])emlrtMxGetData(src);
  for (int32_T i{0}; i < 36; i++) {
    ret[i] = (*r)[i];
  }
  emlrtDestroyArray(&src);
}

static real_T c_emlrt_marshallIn(const mxArray *b_nullptr,
                                 const char_T *identifier)
{
  emlrtMsgIdentifier thisId;
  real_T y;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = c_emlrt_marshallIn(emlrtAlias(b_nullptr), &thisId);
  emlrtDestroyArray(&b_nullptr);
  return y;
}

static real_T c_emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId)
{
  real_T y;
  y = f_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static real_T (*d_emlrt_marshallIn(const mxArray *src,
                                   const emlrtMsgIdentifier *msgId))[3000]
{
  static const int32_T dims[2]{1000, 3};
  real_T(*ret)[3000];
  int32_T iv[2];
  boolean_T bv[2]{false, false};
  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 2U,
                            (const void *)&dims[0], &bv[0], &iv[0]);
  ret = (real_T(*)[3000])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static int32_T (*e_emlrt_marshallIn(const mxArray *src,
                                    const emlrtMsgIdentifier *msgId))[1000]
{
  static const int32_T dims[1]{1000};
  int32_T(*ret)[1000];
  int32_T iv[1];
  boolean_T bv[1]{false};
  emlrtCheckVsBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "int32", false, 1U,
                            (const void *)&dims[0], &bv[0], &iv[0]);
  ret = (int32_T(*)[1000])emlrtMxGetData(src);
  emlrtDestroyArray(&src);
  return ret;
}

static real_T (*emlrt_marshallIn(const mxArray *b_nullptr,
                                 const char_T *identifier))[3000]
{
  emlrtMsgIdentifier thisId;
  real_T(*y)[3000];
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  y = emlrt_marshallIn(emlrtAlias(b_nullptr), &thisId);
  emlrtDestroyArray(&b_nullptr);
  return y;
}

static real_T (*emlrt_marshallIn(const mxArray *u,
                                 const emlrtMsgIdentifier *parentId))[3000]
{
  real_T(*y)[3000];
  y = d_emlrt_marshallIn(emlrtAlias(u), parentId);
  emlrtDestroyArray(&u);
  return y;
}

static void emlrt_marshallIn(const mxArray *b_nullptr, const char_T *identifier,
                             cell_wrap_0 y[5])
{
  emlrtMsgIdentifier thisId;
  thisId.fIdentifier = const_cast<const char_T *>(identifier);
  thisId.fParent = nullptr;
  thisId.bParentIsCell = false;
  emlrt_marshallIn(emlrtAlias(b_nullptr), &thisId, y);
  emlrtDestroyArray(&b_nullptr);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId,
                             cell_wrap_0 y[5])
{
  emlrtMsgIdentifier thisId;
  int32_T iv[2];
  char_T str[11];
  boolean_T bv[2];
  thisId.fParent = parentId;
  thisId.bParentIsCell = true;
  bv[0] = false;
  iv[0] = 1;
  bv[1] = false;
  iv[1] = 5;
  emlrtCheckCell(emlrtRootTLSGlobal, parentId, u, 2U, &iv[0], &bv[0]);
  for (int32_T i{0}; i < 5; i++) {
    emlrtMexSnprintf(&str[0], (size_t)11U, "%d", i + 1);
    thisId.fIdentifier = &str[0];
    emlrt_marshallIn(
        emlrtAlias(emlrtGetCell(emlrtRootTLSGlobal, parentId, u, i)), &thisId,
        y[i].f1);
  }
  emlrtDestroyArray(&u);
}

static void emlrt_marshallIn(const mxArray *u,
                             const emlrtMsgIdentifier *parentId, real_T y[36])
{
  b_emlrt_marshallIn(emlrtAlias(u), parentId, y);
  emlrtDestroyArray(&u);
}

static void emlrt_marshallOut(const real_T u[3000], const mxArray *y)
{
  static const int32_T iv[2]{1000, 3};
  emlrtMxSetData((mxArray *)y, (void *)&u[0]);
  emlrtSetDimensions((mxArray *)y, &iv[0], 2);
}

static real_T f_emlrt_marshallIn(const mxArray *src,
                                 const emlrtMsgIdentifier *msgId)
{
  static const int32_T dims{0};
  real_T ret;
  emlrtCheckBuiltInR2012b(emlrtRootTLSGlobal, msgId, src, "double", false, 0U,
                          (const void *)&dims);
  ret = *static_cast<real_T *>(emlrtMxGetData(src));
  emlrtDestroyArray(&src);
  return ret;
}

void updateParticlesByGroup_api(const mxArray *const prhs[6], int32_T nlhs,
                                const mxArray *plhs[2])
{
  cell_wrap_0 forceMatrices[5];
  const mxArray *prhs_copy_idx_0;
  const mxArray *prhs_copy_idx_1;
  real_T(*pos)[3000];
  real_T(*vel)[3000];
  real_T dt;
  real_T forceLevel;
  int32_T(*ids)[1000];
  prhs_copy_idx_0 = emlrtProtectR2012b(prhs[0], 0, true, -1);
  prhs_copy_idx_1 = emlrtProtectR2012b(prhs[1], 1, true, -1);
  // Marshall function inputs
  pos = emlrt_marshallIn(emlrtAlias(prhs_copy_idx_0), "pos");
  vel = emlrt_marshallIn(emlrtAlias(prhs_copy_idx_1), "vel");
  ids = b_emlrt_marshallIn(emlrtAlias(prhs[2]), "ids");
  emlrt_marshallIn(emlrtAliasP(prhs[3]), "forceMatrices", forceMatrices);
  dt = c_emlrt_marshallIn(emlrtAliasP(prhs[4]), "dt");
  forceLevel = c_emlrt_marshallIn(emlrtAliasP(prhs[5]), "forceLevel");
  // Invoke the target function
  updateParticlesByGroup(*pos, *vel, *ids, forceMatrices, dt, forceLevel);
  // Marshall function outputs
  emlrt_marshallOut(*pos, prhs_copy_idx_0);
  plhs[0] = prhs_copy_idx_0;
  if (nlhs > 1) {
    emlrt_marshallOut(*vel, prhs_copy_idx_1);
    plhs[1] = prhs_copy_idx_1;
  }
}

// End of code generation (_coder_updateParticlesByGroup_api.cu)
