//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// _coder_updateParticlesByGroup_mex.cu
//
// Code generation for function '_coder_updateParticlesByGroup_mex'
//

// Include files
#include "_coder_updateParticlesByGroup_mex.h"
#include "_coder_updateParticlesByGroup_api.h"
#include "rt_nonfinite.h"
#include "updateParticlesByGroup_data.h"
#include "updateParticlesByGroup_initialize.h"
#include "updateParticlesByGroup_terminate.h"
#include <stdexcept>

void emlrtExceptionBridge();
void emlrtExceptionBridge()
{
  throw std::runtime_error("");
}
// Function Definitions
void mexFunction(int32_T nlhs, mxArray *plhs[], int32_T nrhs,
                 const mxArray *prhs[])
{
  mexAtExit(&updateParticlesByGroup_atexit);
  // Module initialization.
  updateParticlesByGroup_initialize();
  try { // Dispatch the entry-point.
    unsafe_updateParticlesByGroup_mexFunction(nlhs, plhs, nrhs, prhs);
    // Module termination.
    updateParticlesByGroup_terminate();
  } catch (...) {
    emlrtCleanupOnException((emlrtCTX *)emlrtRootTLSGlobal);
    throw;
  }
}

emlrtCTX mexFunctionCreateRootTLS()
{
  emlrtCreateRootTLSR2022a(&emlrtRootTLSGlobal, &emlrtContextGlobal, nullptr, 1,
                           (void *)&emlrtExceptionBridge, "windows-1252", true);
  return emlrtRootTLSGlobal;
}

void unsafe_updateParticlesByGroup_mexFunction(int32_T nlhs, mxArray *plhs[2],
                                               int32_T nrhs,
                                               const mxArray *prhs[6])
{
  const mxArray *b_prhs[6];
  const mxArray *outputs[2];
  int32_T b;
  // Check for proper number of arguments.
  if (nrhs != 6) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal, "EMLRT:runTime:WrongNumberOfInputs",
                        5, 12, 6, 4, 22, "updateParticlesByGroup");
  }
  if (nlhs > 2) {
    emlrtErrMsgIdAndTxt(emlrtRootTLSGlobal,
                        "EMLRT:runTime:TooManyOutputArguments", 3, 4, 22,
                        "updateParticlesByGroup");
  }
  // Call the function.
  for (int32_T c{0}; c < 6; c++) {
    b_prhs[c] = prhs[c];
  }
  updateParticlesByGroup_api(b_prhs, nlhs, outputs);
  // Copy over outputs to the caller.
  if (nlhs < 1) {
    b = 1;
  } else {
    b = nlhs;
  }
  emlrtReturnArrays(b, &plhs[0], &outputs[0]);
}

// End of code generation (_coder_updateParticlesByGroup_mex.cu)
