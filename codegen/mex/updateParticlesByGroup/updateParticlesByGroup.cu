#include "hip/hip_runtime.h"
//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// updateParticlesByGroup.cu
//
// Code generation for function 'updateParticlesByGroup'
//

// Include files
#include "updateParticlesByGroup.h"
#include "rt_nonfinite.h"
#include "updateParticlesByGroup_data.h"
#include "updateParticlesByGroup_types.h"
#include "MWCudaDimUtility.hpp"
#include "MWCudaMemoryFunctions.hpp"
#include "stdio.h"
#include "stdlib.h"
#include "string.h"

// Function Declarations
static void checkCudaError(hipError_t errCode, const char_T *file,
                           uint32_T b_line);

static void raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                           const char_T *errorName, const char_T *errorString);

static __global__ void
updateParticlesByGroup_kernel1(const cell_wrap_0 forceMatrices[5],
                               const int32_T ids[1000], const real_T forceLevel,
                               const real_T vel[3000], const real_T pos[3000],
                               real_T forces[3000]);

static __global__ void updateParticlesByGroup_kernel2(const real_T dt,
                                                      const real_T forces[3000],
                                                      real_T pos[3000],
                                                      real_T vel[3000]);

// Function Definitions
static void checkCudaError(hipError_t errCode, const char_T *file,
                           uint32_T b_line)
{
  if (errCode != hipSuccess) {
    raiseCudaError(errCode, file, b_line, hipGetErrorName(errCode),
                   hipGetErrorString(errCode));
  }
}

static void raiseCudaError(int32_T errCode, const char_T *file, uint32_T b_line,
                           const char_T *errorName, const char_T *errorString)
{
  emlrtRTEInfo rtInfo;
  uint64_T len;
  char_T *brk;
  char_T *fn;
  char_T *pn;
  len = strlen(file);
  pn = static_cast<char_T *>(calloc(len + 1ULL, 1ULL));
  fn = static_cast<char_T *>(calloc(len + 1ULL, 1ULL));
  memcpy(pn, file, len);
  memcpy(fn, file, len);
  brk = strrchr(fn, '.');
  *brk = '\x00';
  brk = strrchr(fn, '/');
  if (brk == nullptr) {
    brk = strrchr(fn, '\\');
  }
  if (brk == nullptr) {
    brk = fn;
  } else {
    brk++;
  }
  rtInfo.lineNo = static_cast<int32_T>(b_line);
  rtInfo.colNo = 0;
  rtInfo.fName = brk;
  rtInfo.pName = pn;
  emlrtCUDAError(static_cast<uint32_T>(errCode), (char_T *)errorName,
                 (char_T *)errorString, &rtInfo, emlrtRootTLSGlobal);
}

static __global__ __launch_bounds__(128, 1) void updateParticlesByGroup_kernel1(
    const cell_wrap_0 forceMatrices[5], const int32_T ids[1000],
    const real_T forceLevel, const real_T vel[3000], const real_T pos[3000],
    real_T forces[3000])
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 1000) {
    real_T b_forceMatrices[6];
    real_T b_pos[6];
    //  This function is intended for GPU execution
    //  Marking it for GPU code generation
    //  Update each particle based on its group's force matrix
    b_pos[0] = pos[i];
    b_pos[3] = vel[i];
    b_pos[1] = pos[i + 1000];
    b_pos[4] = vel[i + 1000];
    b_pos[2] = pos[i + 2000];
    b_pos[5] = vel[i + 2000];
    for (int32_T i1{0}; i1 < 6; i1++) {
      real_T d;
      d = 0.0;
      for (int32_T b_i{0}; b_i < 6; b_i++) {
        d += forceMatrices[ids[i] - 1].f1[i1 + 6 * b_i] * b_pos[b_i];
      }
      b_forceMatrices[i1] = d;
    }
    forces[i] = forceLevel * b_forceMatrices[0];
    forces[i + 1000] = forceLevel * b_forceMatrices[1];
    forces[i + 2000] = forceLevel * b_forceMatrices[2];
  }
}

static __global__ __launch_bounds__(128, 1) void updateParticlesByGroup_kernel2(
    const real_T dt, const real_T forces[3000], real_T pos[3000],
    real_T vel[3000])
{
  int32_T i;
  i = static_cast<int32_T>(mwGetGlobalThreadIndex());
  if (i < 3000) {
    real_T d;
    //  Update velocities and positions
    d = vel[i] + forces[i] * dt;
    vel[i] = d;
    pos[i] += d * dt;
  }
}

void updateParticlesByGroup(real_T cpu_pos[3000], real_T cpu_vel[3000],
                            const int32_T cpu_ids[1000],
                            const cell_wrap_0 cpu_forceMatrices[5], real_T dt,
                            real_T forceLevel)
{
  cell_wrap_0(*gpu_forceMatrices)[5];
  real_T(*gpu_forces)[3000];
  real_T(*gpu_pos)[3000];
  real_T(*gpu_vel)[3000];
  int32_T(*gpu_ids)[1000];
  checkCudaError(mwCudaMalloc(&gpu_forces, 24000ULL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_forceMatrices, 1440ULL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_ids, 4000ULL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_vel, 24000ULL), __FILE__, __LINE__);
  checkCudaError(mwCudaMalloc(&gpu_pos, 24000ULL), __FILE__, __LINE__);
  //  This function is intended for GPU execution
  //  Marking it for GPU code generation
  //  Update each particle based on its group's force matrix
  checkCudaError(hipMemcpy(*gpu_forceMatrices, cpu_forceMatrices, 1440ULL,
                            hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  checkCudaError(hipMemcpy(*gpu_ids, cpu_ids, 4000ULL, hipMemcpyHostToDevice),
                 __FILE__, __LINE__);
  checkCudaError(
      hipMemcpy(*gpu_vel, cpu_vel, 24000ULL, hipMemcpyHostToDevice), __FILE__,
      __LINE__);
  checkCudaError(
      hipMemcpy(*gpu_pos, cpu_pos, 24000ULL, hipMemcpyHostToDevice), __FILE__,
      __LINE__);
  updateParticlesByGroup_kernel1<<<dim3(8U, 1U, 1U), dim3(128U, 1U, 1U)>>>(
      *gpu_forceMatrices, *gpu_ids, forceLevel, *gpu_vel, *gpu_pos,
      *gpu_forces);
  //  Update velocities and positions
  updateParticlesByGroup_kernel2<<<dim3(24U, 1U, 1U), dim3(128U, 1U, 1U)>>>(
      dt, *gpu_forces, *gpu_pos, *gpu_vel);
  checkCudaError(
      hipMemcpy(cpu_pos, *gpu_pos, 24000ULL, hipMemcpyDeviceToHost), __FILE__,
      __LINE__);
  checkCudaError(
      hipMemcpy(cpu_vel, *gpu_vel, 24000ULL, hipMemcpyDeviceToHost), __FILE__,
      __LINE__);
  checkCudaError(mwCudaFree(*gpu_pos), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_vel), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_ids), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_forceMatrices), __FILE__, __LINE__);
  checkCudaError(mwCudaFree(*gpu_forces), __FILE__, __LINE__);
}

// End of code generation (updateParticlesByGroup.cu)
