//
// Academic License - for use in teaching, academic research, and meeting
// course requirements at degree granting institutions only.  Not for
// government, commercial, or other organizational use.
//
// updateParticlesByGroup_initialize.cu
//
// Code generation for function 'updateParticlesByGroup_initialize'
//

// Include files
#include "updateParticlesByGroup_initialize.h"
#include "_coder_updateParticlesByGroup_mex.h"
#include "rt_nonfinite.h"
#include "updateParticlesByGroup_data.h"

// Function Definitions
void updateParticlesByGroup_initialize()
{
  mex_InitInfAndNan();
  emlrtInitGPU(emlrtRootTLSGlobal);
  hipGetLastError();
  mexFunctionCreateRootTLS();
  emlrtClearAllocCountR2012b(emlrtRootTLSGlobal, false, 0U, nullptr);
  emlrtEnterRtStackR2012b(emlrtRootTLSGlobal);
  emlrtLicenseCheckR2022a(emlrtRootTLSGlobal,
                          "EMLRT:runTime:MexFunctionNeedsLicense",
                          "distrib_computing_toolbox", 2);
  emlrtFirstTimeR2012b(emlrtRootTLSGlobal);
}

// End of code generation (updateParticlesByGroup_initialize.cu)
